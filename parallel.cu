#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;

#define PI 3.14159265358979323846
#define MAX_RADII 50
#define Kn 9.9
#define THRESH 1000

typedef pair<int,int> pii;

typedef struct Pixel {
    uint8_t r,g,b;
} Pixel;

typedef struct Image {
    int w, h;
    Pixel** img;
    Image(int width, int height) {
        w = width; 
        h = height;
        img = (Pixel**)calloc(h, sizeof(Pixel*) * h);
        for(int i = 0; i < h ;i++){
            img[i] = (Pixel*)calloc(w, sizeof(Pixel));
        }
    }
} Image;


__global__ void grayscaleKernel(Pixel* img, double* grayscaleImg, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        Pixel pixel = img[idx];
        grayscaleImg[idx] = 0.2125 * pixel.r + 0.7154 * pixel.g + 0.0721 * pixel.b;
    }
}


void gradient(double** grayscaleImg, double** gradients, double** gradientDir, Image* color,
              double** gradX, double** gradY){
    double xSobel[3][3] = {
        {1, 0, -1},
        {2, 0, -2},
        {1, 0, -1},
    };
    double ySobel[3][3] = {
        {1, 2, 1},
        {0, 0, 0},
        {-1, -2, -1},
    };

    for(int i = 1; i < color->h - 1; i++){
        for(int j = 1; j < color->w - 1; j++){
            // printf("%d, %d\n", i, j);
            double xg, yg;
            xg = 0;
            yg = 0;
            for(int dy = 0; dy < 3; dy++){
                for(int dx = 0; dx < 3; dx++){
                    xg += grayscaleImg[dy+i-1][dx+j-1] * xSobel[dy][dx];
                    yg += grayscaleImg[dy+i-1][dx+j-1] * ySobel[dy][dx];
                }
            }
            double mag = sqrt(xg*xg + yg*yg);
            gradients[i][j] = (mag > 200) ? mag : 0;
            gradY[i][j] = yg; gradX[i][j] = xg;
// ")
            // if(gradients[i][j]){
            //     gradientDir[i][j] = atan(yg/xg);
            //     if(isnan(gradientDir[i][j])) gradientDir[i][j] = PI / 2 * ((yg > 0) - (yg < 0));
            // }
            // else {
            //     gradientDir[i][j] = -10;
            // }
            gradientDir[i][j] = atan(yg/xg);
            if(isnan(gradientDir[i][j])) gradientDir[i][j] = PI / 2 * ((yg > 0) - (yg < 0));
        }
    }
}

__global__ void gradientKernel(double** grayscaleImg, double** gradients, double** gradX, double** gradY, double** gradientDir, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        double xSobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
        double ySobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};
        double xg = 0, yg = 0;

        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                double pixel = grayscaleImg[(y + dy) * width + (x + dx)];
                xg += pixel * xSobel[dy + 1][dx + 1];
                yg += pixel * ySobel[dy + 1][dx + 1];
            }
        }
        double mag = sqrt(xg * xg + yg * yg);
        gradients[y][x] = (mag > 200) ? mag : 0;
        gradX[y][x] = xg;
        gradY[y][x] = yg;
        double angle = atan2(yg, xg);
        if(isnan(gradientDir[y][x])) gradientDir[y][x] = PI / 2 * ((yg > 0) - (yg < 0));
    }
}

__global__ void gradientKernel(double* grayscaleImg, double* gradients, double* gradX, double* gradY, double* gradientDir, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        double xSobel[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
        double ySobel[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};
        double xg = 0, yg = 0;

        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                double pixel = grayscaleImg[(y + dy) * width + (x + dx)];
                xg += pixel * xSobel[dy + 1][dx + 1];
                yg += pixel * ySobel[dy + 1][dx + 1];
            }
        }
        double mag = sqrt(xg * xg + yg * yg);
        gradients[y * width + x] = (mag > 200) ? mag : 0;
        gradX[y * width + x] = xg;
        gradY[y * width + x] = yg;
        double angle = atan2(yg, xg);
        if(isnan(gradientDir[y * width + x])) gradientDir[y * width + x] = PI / 2 * ((yg > 0) - (yg < 0));
    }
}

__global__ void NMSKernel(double **gradients, double **gradientDir, double **nms_gradients, int width, int height, double PI) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        int idx = y * width + x;
        double n1, n2;
        if (gradients[y][x] != 0) {
            double dir = gradientDir[y][x];
            if (dir > PI / 4 || dir < -PI / 4) {
                n1 = gradients[y - 1][x];
                n2 = gradients[y + 1][x];
            } else if (dir > 0) {
                n1 = gradients[y - 1][x + 1];
                n2 = gradients[y - 1][x - 1];
            } else if (dir > -PI / 4) {
                n1 = gradients[y + 1][x + 1];
                n2 = gradients[y - 1][x - 1];
            } else {
                n1 = 0;
                n2 = 0;
            }

            if (gradients[y][x] >= n1 && gradients[y][x] >= n2) {
                nms_gradients[y][x] = gradients[y][x];
            } else {
                nms_gradients[y][x] = 0;
            }
        }
    }
}

__global__ void NMSKernel(double *gradients, double *gradientDir, double *nms_gradients, int width, int height, double PI) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        int idx = y * width + x;
        double n1, n2;
        if (gradients[idx] != 0) {
            double dir = gradientDir[idx];
            if (dir > PI / 4 || dir < -PI / 4) {
                n1 = gradients[(y - 1) * width + x];
                n2 = gradients[(y + 1) * width + x];
            } else if (dir > 0) {
                n1 = gradients[(y - 1) * width + (x + 1)];
                n2 = gradients[(y + 1) * width + (x - 1)];
            } else if (dir > -PI / 4) {
                n1 = gradients[(y + 1) * width + (x + 1)];
                n2 = gradients[(y - 1) * width + (x - 1)];
            } else {
                n1 = 0;
                n2 = 0;
            }

            if (gradients[idx] >= n1 && gradients[idx] >= n2) {
                nms_gradients[idx] = gradients[idx];
            } else {
                nms_gradients[idx] = 0;
            }
        }
    }
}

__global__ void radialSymmetryKernel(double *gradX, double *gradY, double *gradients, double *O, double *M, int width, int height, int max_radii) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height && gradients[y * width + x] != 0) {
        double dy = gradY[y * width + x] / gradients[y * width + x];
        double dx = gradX[y * width + x] / gradients[y * width + x];

        for (int r = 1; r < max_radii; r++) {
            int p_plus_y = y + dy * r;
            int p_plus_x = x + dx * r;
            int p_minus_y = y - dy * r;
            int p_minus_x = x - dx * r;

            if (p_plus_y >= 0 && p_plus_y < height && p_plus_x >= 0 && p_plus_x < width) {
                atomicAdd(&O[r * height * width + p_plus_y * width + p_plus_x], 1);
                atomicAdd(&M[r * height * width + p_plus_y * width + p_plus_x], gradients[y * width + x]);
            }
            if (p_minus_y >= 0 && p_minus_y < height && p_minus_x >= 0 && p_minus_x < width) {
                atomicAdd(&O[r * height * width + p_minus_y * width + p_minus_x], -1);
                atomicAdd(&M[r * height * width + p_minus_y * width + p_minus_x], -gradients[y * width + x]);
            }
        }
    }
}


//gaussian convolve
__global__ void gaussConvolve(double ***M, double **postGauss, int **radii, Image *color){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    double gauss[3][3] = {
        {1.0/16.0, 2.0/16.0, 1.0/16.0},
        {2.0/16.0, 4.0/16.0, 2.0/16.0},
        {1.0/16.0, 2.0/16.0, 1.0/16.0}
    };

    // for(int i = 0; i < 3; i++){
    //         for(int j = 0; j < 3; j++){
    //             gauss[i][j] =  1/16 * gauss[i][j];
    //     }
    // }

    for(int i = 0; i < color->h; i++){
        for(int j = 0; j < color->w; j++){
            double t = 0;
            int best_r = 0;
            for(int r = 0; r < MAX_RADII; r++){
                // t = max(M[r][i][j], t);
                if(M[r][i][j] > t){
                    best_r = r;
                    t = M[r][i][j];
                }
            }
            radii[i][j] = best_r;
            M[0][i][j] = t;
        }
    }
    for(int i = 1; i < color->h - 1; i++){
        for(int j = 1; j < color->w - 1; j++){
            // printf("(%d, %d)", i, j);
            double g = 0;
            for(int dy = 0; dy < 3; dy++){
                for(int dx = 0; dx < 3; dx++){
                    // printf("(%d, %d), %f || ", dy, dx, gauss[dy][dx]);
                    g += M[0][dy+i-1][dx+j-1] * gauss[dy][dx];
                    if(g != 0){
                        // printf("(%d,%d) %f, %f\n", i, j, M[0][dy+i-1][dx+j-1], gauss[dy][dx]);
                    }
                }
            }
            // printf("\n");
            // printf("(%d,%d)->%f\n", i,j, g);
            postGauss[i][j] = (g > THRESH) ? g : 0;
        }
    }
}

void postGaussNMS(double **postGauss, double **gaussNMS, int **radii,Image* color){
    for(int i = 1; i < color->h -1; i++){
        for(int j = 1; j < color->w -1; j++){
            double maxNeighbor = 0.0;
            for(int dy = -1; dy < 2; dy++){
                for(int dx = -1; dx < 2; dx++){
                    if(dy != 0 || dx != 0){
                        if(maxNeighbor < postGauss[i+dy][j+dx]){
                            maxNeighbor = postGauss[i+dy][j+dx];
                        }
                        // maxNeighbor = max(maxNeighbor, postGauss[i + dy][j + dx], maxNeighbor);
                    }
                }
            }
            if(postGauss[i][j] >= maxNeighbor){
                gaussNMS[i][j] = postGauss[i][j];
            }
            else {
                radii[i][j] = 0;
            }
        }
    }
}